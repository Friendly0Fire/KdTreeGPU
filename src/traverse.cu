#include "hip/hip_runtime.h"
#include "DevBetterDipole.h"

namespace cukd {
namespace device {

template<typename Point>
struct KdTreeTraverseData {
    IndexedPtr<PreorderTreeOpaqueElement, PreorderTreeOpaqueIndex> preorder;
    typename Point::RawArray elements;
    typename Point::RawArray avgElements;

    NodeDepth maxDepth;

    __device__
    KdTreeTraverseData(IndexedPtr<PreorderTreeOpaqueElement, PreorderTreeOpaqueIndex> preorder,
                       const typename Point::RawArray& elements,
                       const typename Point::RawArray& avgElements,
                       NodeDepth maxDepth)
        : preorder(preorder), elements(elements), avgElements(avgElements), maxDepth(maxDepth) {}
};

using KdTreeTraverseDataIP = KdTreeTraverseData<IlluminationPoint>;

struct KdTreeTraverseDataSP : KdTreeTraverseData<ShadingPoint> {

    IndexedPtr<ElementCountAndIsSmall, NodeIndex> sizeAndIsSmall;
    IndexedPtr<PixelIndex, NodeIndex> pixelOffsets;
    IndexedPtr<PackedPixel, PixelIndex> pixels;
    
    __device__
    KdTreeTraverseDataSP(IndexedPtr<PreorderTreeOpaqueElement, PreorderTreeOpaqueIndex> preorder,
                         const ShadingPoint::RawArray& elements,
                         const ShadingPoint::RawArray& avgElements,
                         IndexedPtr<ElementCountAndIsSmall, NodeIndex> sizeAndIsSmall,
                         IndexedPtr<PixelIndex, NodeIndex> pixelOffsets,
                         IndexedPtr<PackedPixel, PixelIndex> pixels,
                         NodeDepth maxDepth)
        : KdTreeTraverseData(preorder, elements, avgElements, maxDepth),
          sizeAndIsSmall(sizeAndIsSmall), pixelOffsets(pixelOffsets), pixels(pixels) {}
};

__device__
inline void updateNodeAABB(const PreorderInnerNode* node, UAABB& box, bool goingRight) {
    if(node->isLeaf())
        return;

    if(goingRight)
        sliceT(node->splitAxis(), box.minimum) = node->splitPosition();
    else
        sliceT(node->splitAxis(), box.maximum) = node->splitPosition();
}

__device__
inline bool overlaps(const UAABB& a, const UAABB& b) {
    return interval_overlap(a.minimum.vec.x, a.maximum.vec.x, b.minimum.vec.x, b.maximum.vec.x)
        && interval_overlap(a.minimum.vec.y, a.maximum.vec.y, b.minimum.vec.y, b.maximum.vec.y)
        && interval_overlap(a.minimum.vec.z, a.maximum.vec.z, b.minimum.vec.z, b.maximum.vec.z);
}

__device__
inline void shadeNodes(const DevSubsurface* ss, float4* output, int outputPitch,
                       const PreorderNode* shadeNode, const PreorderNode* illumNode,
                       const KdTreeTraverseDataSP* shade,
                       const KdTreeTraverseDataIP* illum) {
    const UFloat4& shadePosition = shade->avgElements.positions[shadeNode->elementBackReference()];
    const float3& shadeNormal = shade->avgElements.normals[shadeNode->elementBackReference()];
    const float3& shadeDirection = shade->avgElements.directions[shadeNode->elementBackReference()];

    const UFloat4& illumPosition = illum->avgElements.positions[illumNode->elementBackReference()];
    const float3& illumIrradiance = illum->avgElements.irradiances[illumNode->elementBackReference()];
    const float& illumArea = illum->avgElements.areas[illumNode->elementBackReference()];
    float3 contrib = ss->query(shadePosition, shadeNormal, shadeDirection,
                     illumPosition, illumIrradiance, illumArea);

    PixelIndex pixelsStart = shade->pixelOffsets[shadeNode->backReference()];
    PixelCount pixelsCount = shadeNode->pixelCount(shade->sizeAndIsSmall);

    for(PixelIndex i = pixelsStart; i <= pixelsStart + pixelsCount; ++i) {
        int2 px = int2(shade->pixels[i]);
        float4* outputElement = (float4*)((char*)output + px.y * outputPitch) + px.x;
        *outputElement += make_float4(contrib, 0.f);
    }
}

__global__
void traverseSubtreeDual(const DevSubsurface* ss, float4* output, int outputPitch,
                         const KdTreeTraverseDataSP* shade, const KdTreeTraverseDataIP* illum,
                         float threshold, NodeDepth depth,
                         PreorderTreeOpaqueIndex baseShadeNodeIdx, PreorderTreeOpaqueIndex baseIllumNodeIdx,
                         UAABB shadeBox, UAABB illumBox) {
    int shadeMask = threadIdx.x;
    int illumMask = threadIdx.y;
    
    PreorderTreeOpaqueIndex shadeNodeIdx = baseShadeNodeIdx;
    PreorderTreeOpaqueIndex illumNodeIdx = baseIllumNodeIdx;

    for(int currentMask = 1; currentMask <= kDualTreeChunkDepth; currentMask <<= 1, ++depth) {
    
        const auto shadeNode = nodeT(shade->preorder[shadeNodeIdx]);
        const auto illumNode = nodeT(illum->preorder[illumNodeIdx]);

        bool shadeGoingRight = (shadeMask & currentMask) != 0;
        bool illumGoingRight = (illumMask & currentMask) != 0;
        updateNodeAABB(shadeNode->asInner(), shadeBox, shadeGoingRight);
        updateNodeAABB(illumNode->asInner(), illumBox, illumGoingRight);

        if(shadeNode->isLeaf() && illumNode->isLeaf()) {
            shadeNodes(ss, output, outputPitch, shadeNode, illumNode, shade, illum);
            return;
        }

        float irrArea = illum->avgElements.areas[illumNode->elementBackReference()];
        float shadeArea = threshold * squared_distance_ufloat4(shade->avgElements.positions[shadeNode->elementBackReference()], illum->avgElements.positions[illumNode->elementBackReference()]);

        if(irrArea < shadeArea && overlaps(shadeBox, illumBox)) {
            shadeNodes(ss, output, outputPitch, shadeNode, illumNode, shade, illum);
            return;
        }

        if(!illumNode->isLeaf())
            illumNodeIdx = illumGoingRight ? illumNode->asInner()->rightIndex() : illumNodeIdx + 3_ptoi;
        if(!shadeNode->isLeaf())
            shadeNodeIdx = shadeGoingRight ? shadeNode->asInner()->rightIndex() : shadeNodeIdx + 3_ptoi;
    }

    // We know for a fact at least one node is not a leaf if we reached this point
    dim3 grid(1,1,1);
    dim3 blocks(min(int(shade->maxDepth - depth), 1 << kDualTreeChunkDepth),min(int(illum->maxDepth - depth), 1 << kDualTreeChunkDepth),1);
    traverseSubtreeDual CU_OPT(grid,blocks)(ss, output, outputPitch, shade, illum, threshold, depth, shadeNodeIdx, illumNodeIdx, shadeBox, illumBox);
}

}


KdTreeTraverser::KdTreeTraverser(std::shared_ptr<Subsurface> subsurface, std::shared_ptr<KdTree<ShadingPoint>> shadingTree, std::shared_ptr<KdTree<IlluminationPoint>> illuminationTree)
    : m_subsurface(subsurface), m_shadeTree(shadingTree), m_illumTree(illuminationTree) { }

void KdTreeTraverser::traverse(float4* outputMemory, const uint3& dims) {
    // Spawn NxN blocks, with N a power of two
    // Each bit of N marks whether the thread should walk down the left or right child for their respective tree
    // When the thread reaches a halting threshold, it performs the relevant computation
    // When the thread reaches its maximum depth (i.e. log_2 N), it spawns another block for its node
    // Should a leaf be reached, only the thread with zeroes for everything after the leaf's bit mask (i.e. the one
    //  which would continue down left/left children only) continues, the others return
    // Start with 1 block and go from there
    // 16x16 blocks are probably ideal

    DevObject<device::KdTreeTraverseDataSP> shade;
    shade.makeDevice(m_shadeTree->m_preorderTree.pointer(), m_shadeTree->m_KDTreeNWA.m_points.rawArray(), m_shadeTree->m_preorderInnerTreeElements.rawArray(), m_shadeTree->m_preorderBandwidthSizeAndIsSmall.pointer(), m_shadeTree->m_preorderBandwidthOffsets.pointer(), m_shadeTree->m_preorderPixels.pointer(), m_shadeTree->maxDepth());

    DevObject<device::KdTreeTraverseDataIP> illum;
    illum.makeDevice(m_illumTree->m_preorderTree.pointer(), m_illumTree->m_KDTreeNWA.m_points.rawArray(), m_illumTree->m_preorderInnerTreeElements.rawArray(), m_illumTree->maxDepth());

    UAABB shadeBox = m_shadeTree->m_rootAABB, illumBox = m_illumTree->m_rootAABB;
    
    auto ss = DevBetterDipole::create(static_cast<BetterDipole*>(m_subsurface.get()));

    hipMemset2D(outputMemory, dims.z, 0, dims.x, dims.y);

    device::traverseSubtreeDual CU_OPT(1,1)(ss.devPointer(), outputMemory, dims.z,
                                                 shade.devPointer(), illum.devPointer(), m_threshold, 0, 0, 0,
                                          shadeBox, illumBox);
    reportCudaErrorsForFunction("traverseSubtreeDual failed");
}



}