﻿#include "Gpu.h"
#include "KdNode.h"

/* Create a simple k-d tree and print its topology for inspection. */
sint main(sint argc, char **argv)
{
	// Set the defaults then parse the input arguments.
	sint numPoints = 4194304;
	sint extraPoints = 100;
	sint numDimensions = 3;
	sint numThreads = 512;
	sint numBlocks = 32;
	sint searchDistance = 20000000;
	sint maximumNumberOfNodesToPrint = 5;

	for (sint i = 1; i < argc; i++) {
		if ( 0 == strcmp(argv[i], "-n") || 0 == strcmp(argv[i], "--numPoints") ) {
			numPoints = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-x") || 0 == strcmp(argv[i], "--extraPoints") ) {
			extraPoints = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-d") || 0 == strcmp(argv[i], "--numDimensions") ) {
			numDimensions = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-t") || 0 == strcmp(argv[i], "--numThreads") ) {
			numThreads = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-b") || 0 == strcmp(argv[i], "--numBlocks") ) {
			numBlocks = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-s") || 0 == strcmp(argv[i], "--searchDistance") ) {
			searchDistance = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-p") || 0 == strcmp(argv[i], "--maximumNodesToPrint") ) {
			maximumNumberOfNodesToPrint = atol(argv[++i]);
			continue;
		}
        std::cout << "Unsupported command-line argument: " <<  argv[i] << std::endl;
		exit(1);
	}

	sint  i = maximumNumberOfNodesToPrint + numDimensions + extraPoints;
	// Declare the two-dimensional coordinates array that contains (x,y,z) coordinates.
	/*
    sint coordinates[NUM_TUPLES][DIMENSIONS] = {
    {2,3,3}, {5,4,2}, {9,6,7}, {4,7,9}, {8,1,5},
    {7,2,6}, {9,4,1}, {8,4,2}, {9,7,8}, {6,3,1},
    {3,4,5}, {1,6,8}, {9,5,3}, {2,1,3}, {8,7,6},
    {5,4,2}, {6,3,1}, {8,7,6}, {9,6,7}, {2,1,3},
    {7,2,6}, {4,7,9}, {1,6,8}, {3,4,5}, {9,4,1} };
	 */
	//  gpu = new Gpu(numThreads,numBlocks,0,numDimensions);
	Gpu::gpuSetup(numThreads, numBlocks, numDimensions);
	if (Gpu::getNumThreads() == 0 || Gpu::getNumBlocks() == 0) {
		cout << "KdNode Tree cannot be built with " << numThreads << " threads or " << numBlocks << " blocks." << endl;
		exit(1);
	}
	cout << "Points = " << numPoints << " dimensions = " << numDimensions << ", threads = " << numThreads << ", blocks = " << numBlocks << endl;

	srand(0);
	KdCoord (*coordinates) = new KdCoord[numPoints*numDimensions];
	for ( i = 0; i<numPoints; i++) {
		for (sint j=0; j<numDimensions; j++) {
			coordinates[i*numDimensions+j] = (KdCoord)rand();
			//coordinates[i*numDimensions+j] = (j==1)? (numPoints-i) : i;
			//coordinates[i*numDimensions+j] =  i;
		}
	}

	// Create the k-d tree.  First copy the data to a tuple in its kdNode.
	// also null out the gt and lt references
	// create and initialize the kdNodes array
	KdNode *kdNodes = new KdNode[numPoints];
	if (kdNodes == NULL) {
		printf("Can't allocate %d kdNodes\n", numPoints);
		exit (1);
	}

	KdNode *root = KdNode::createKdTree(kdNodes, coordinates, numDimensions, numPoints);

	// Print the k-d tree "sideways" with the root at the left.
	cout << endl;

	if (searchDistance == 0){
		return 0;
	}
	TIMER_DECLARATION();
	// Search the k-d tree for the k-d nodes that lie within the cutoff distance of the first tuple.
	KdCoord* query = (KdCoord *)malloc(numDimensions * sizeof(KdCoord));
	for (sint i = 0; i < numDimensions; i++) {
		query[i] = coordinates[i];
	}
	// read the KdTree back from GPU
	Gpu::getKdTreeResults(kdNodes, numPoints);
#define VERIFY_ON_HOST
#ifdef VERIFY_ON_HOST
	sint numberOfNodes = root->verifyKdTree( kdNodes, coordinates, numDimensions, 0);
	cout <<  "Number of nodes on host = " << numberOfNodes << endl;
#endif
	TIMER_START();
	list<KdNode> kdList = root->searchKdTree(kdNodes, coordinates, query, searchDistance, numDimensions, 0);
	TIMER_STOP(double searchTime);
	cout << "searchTime = " << fixed << setprecision(2) << searchTime << " seconds" << endl << endl;

	cout << endl << kdList.size() << " nodes within " << searchDistance << " units of ";
	KdNode::printTuple(query, numDimensions);
	cout << " in all dimensions." << endl << endl;
	if (kdList.size() != 0) {
		cout << "List of k-d nodes within " << searchDistance << "-unit search distance follows:" << endl << endl;
		list<KdNode>::iterator it;
		for (it = kdList.begin(); it != kdList.end(); it++) {
			KdNode::printTuple(coordinates+it->getTuple()*numDimensions, numDimensions);
			cout << " ";
		}
		cout << endl;
	}
	return 0;
}
